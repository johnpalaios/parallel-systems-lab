#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "kmeans.h"
#include "alloc.h"
#include "error.h"

#ifdef __HIPCC__
inline void checkCuda(hipError_t e) {
    if (e != hipSuccess) {
        // hipGetErrorString() isn't always very helpful. Look up the error
        // number in the hipError_t enum in hip/driver_types.h in the CUDA includes
        // directory for a better explanation.
        error("CUDA Error %d: %s\n", e, hipGetErrorString(e));
    }
}

inline void checkLastCudaError() {
    checkCuda(hipGetLastError());
}
#endif

__device__ int get_tid(){
	return threadIdx.x + blockIdx.x * blockDim.x; /* TODO: copy me from naive version... */
}

/* square of Euclid distance between two multi-dimensional points using column-base format */
__host__ __device__ inline static
double euclid_dist_2_transpose(int numCoords,
                    int    numObjs,
                    int    numClusters,
                    double *objects,     // [numCoords][numObjs]
                    double *clusters,    // [numCoords][numClusters]
                    int    objectId,
                    int    clusterId)
{
    int i;
    double ans=0.0;

	/* TODO: Calculate the euclid_dist of elem=objectId of objects from elem=clusterId from clusters, but for column-base format!!! */
	/* NAIVE :
	for(i = 0; i < numCoords; i++) {
    	ans += (objects[objectId*numCoords+i] - clusters[clusterId*numCoords+i]) * (objects[objectId*numCoords+i] - clusters[clusterId*numCoords+i]);
    }	
	*/

	for(i = 0; i < numCoords; i++) {
		ans += (objects[objectId+(numObjs*i)] - clusters[clusterId+(numClusters*i)]) * (objects[objectId+(numObjs*i)] - clusters[clusterId+(numClusters*i)]);
	}
    return(ans);
}

__global__ static
void find_nearest_cluster(int numCoords,
                          int numObjs,
                          int numClusters,
                          double *objects,           //  [numCoords][numObjs]
                          double *deviceClusters,    //  [numCoords][numClusters]
                          int *membership,          //  [numObjs]
                          double *devdelta)
{
	/* TODO: copy me from naive version... */
	/* Get the global ID of the thread. */
    int tid = get_tid();
    /*
	int local_tid = threadIdx.x;
    extern __shared__ double partial_devdelta[];
    partial_devdelta[local_tid] = 0;
	*/
    /* TODO: Maybe something is missing here... should all threads run this? */
    // threads with tid > numObjs ??
    if (tid < numObjs) {
        int   index, i;
        double dist, min_dist;

        /* find the cluster id that has min distance to object */
        index = 0;
        /* TODO: call min_dist = euclid_dist_2(...) with correct objectId/clusterId */
        // objectId is tid maybe ?
        min_dist = euclid_dist_2_transpose(numCoords, numObjs, numClusters, objects, deviceClusters, tid, 0);
        for (i=1; i<numClusters; i++) {
        	/* TODO: call dist = euclid_dist_2(...) with correct objectId/clusterId */
            // again objectId is tid ?
            dist = euclid_dist_2_transpose(numCoords, numObjs, numClusters, objects, deviceClusters, tid, i);
            /* no need square root */
            if (dist < min_dist) { /* find the min and its array index */
            	min_dist = dist;
                index    = i;
            }
        }

        if (membership[tid] != index) {
            /* TODO: Maybe something is missing here... is this write safe? */
	        // reduce cuda variable..how??
            // we will use shared memory for the reduction
            //partial_devdelta[local_tid] += 1.0;
            //(*devdelta)+= 1.0;
	    	atomicAdd(devdelta, 1.0);
        }
        /* assign the deviceMembership to object objectId */
        membership[tid] = index;
    }
	/*
    __syncthreads();
    int i = blockDim.x / 2;
    while(i != 0) {
        if(local_tid < i) {
            partial_devdelta[local_tid] += partial_devdelta[local_tid + i];
        }
        __syncthreads();
        i /= 2;
    }
    if(local_tid == 0) {
        atomicAdd(devdelta, partial_devdelta[0]);
    }	
	*/
}


//
//  ----------------------------------------
//  DATA LAYOUT
//
//  objects         [numObjs][numCoords]
//  clusters        [numClusters][numCoords]
//  dimObjects      [numCoords][numObjs]
//  dimClusters     [numCoords][numClusters]
//  newClusters     [numCoords][numClusters]
//  deviceObjects   [numCoords][numObjs]
//  deviceClusters  [numCoords][numClusters]
//  ----------------------------------------
//
/* return an array of cluster centers of size [numClusters][numCoords]       */            
void kmeans_gpu(	double *objects,      /* in: [numObjs][numCoords] */
		               	int     numCoords,    /* no. features */
		               	int     numObjs,      /* no. objects */
		               	int     numClusters,  /* no. clusters */
		               	double   threshold,    /* % objects change membership */
		               	long    loop_threshold,   /* maximum number of iterations */
		               	int    *membership,   /* out: [numObjs] */
						double * clusters,   /* out: [numClusters][numCoords] */
						int blockSize)  
{
    double timing = wtime(), timing_internal, timer_min = 1e42, timer_max = 0; 
	int    loop_iterations = 0; 
    int      i, j, index, loop=0;
    int     *newClusterSize; /* [numClusters]: no. objects assigned in each
                                new cluster */
    double  delta = 0, *dev_delta_ptr;          /* % of objects change their clusters */
    
    /* TODO: Transpose dims */
    double  **dimObjects = (double**) calloc_2d(numCoords, numObjs, sizeof(double)); //calloc_2d(...) -> [numCoords][numObjs]
    double  **dimClusters = (double**) calloc_2d(numCoords, numClusters, sizeof(double));  //calloc_2d(...) -> [numCoords][numClusters]
    double  **newClusters = (double**) calloc_2d(numCoords, numClusters, sizeof(double));  //calloc_2d(...) -> [numCoords][numClusters]
    
    double *deviceObjects;
    double *deviceClusters;
    int *deviceMembership;

    printf("\n|-----------Transpose GPU Kmeans------------|\n\n");
    
    //  TODO: Copy objects given in [numObjs][numCoords] layout to new
    //  [numCoords][numObjs] layout
    for (i = 0; i < numCoords; i++) {
        for (j = 0; j < numObjs; j++) {
            dimObjects[i][j] = objects[j*numCoords+i];
        }
    }
	
    /* pick first numClusters elements of objects[] as initial cluster centers*/
    for (i = 0; i < numCoords; i++) {
        for (j = 0; j < numClusters; j++) {
            dimClusters[i][j] = dimObjects[i][j];
        }
    }
	
    /* initialize membership[] */
    for (i=0; i<numObjs; i++) membership[i] = -1;

    /* need to initialize newClusterSize and newClusters[0] to all 0 */
    newClusterSize = (int*) calloc(numClusters, sizeof(int));
    assert(newClusterSize != NULL); 
    
    timing = wtime() - timing;
    printf("t_alloc: %lf ms\n\n", 1000*timing);
    timing = wtime(); 

    const unsigned int numThreadsPerClusterBlock = (numObjs > blockSize)? blockSize: numObjs;
    const unsigned int numClusterBlocks = numObjs/blockSize + 1; /* TODO: Calculate Grid size, e.g. number of blocks. */
    //const unsigned int clusterBlockSharedDataSize = numThreadsPerClusterBlock*sizeof(double);;
    const unsigned int clusterBlockSharedDataSize = 0;
 
    checkCuda(hipMalloc(&deviceObjects, numObjs*numCoords*sizeof(double)));
    checkCuda(hipMalloc(&deviceClusters, numClusters*numCoords*sizeof(double)));
    checkCuda(hipMalloc(&deviceMembership, numObjs*sizeof(int)));
    checkCuda(hipMalloc(&dev_delta_ptr, sizeof(double)));
    timing = wtime() - timing;
    printf("t_alloc_gpu: %lf ms\n\n", 1000*timing);
    timing = wtime(); 
    
    checkCuda(hipMemcpy(deviceObjects, dimObjects[0],
              numObjs*numCoords*sizeof(double), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(deviceMembership, membership,
              numObjs*sizeof(int), hipMemcpyHostToDevice));
    timing = wtime() - timing;
    printf("t_get_gpu: %lf ms\n\n", 1000*timing);
    timing = wtime();   
   	double cpu_to_gpu_time, gpu_start, gpu_time,
        gpu_to_cpu_start, gpu_to_cpu_time, cpu_start, cpu_time; 
    do {
    	timing_internal = wtime();

		/* GPU part: calculate new memberships */
		        
        /* TODO: Copy clusters to deviceClusters
        checkCuda(hipMemcpy(...)); */
		// maybe dimClusters[0] in similarity with dimObjects[0] above ?
        checkCuda(hipMemcpy(deviceClusters, dimClusters[0],
				  numClusters*numCoords*sizeof(double), hipMemcpyHostToDevice));
        checkCuda(hipMemset(dev_delta_ptr, 0, sizeof(double)));          
		
		cpu_to_gpu_time = wtime() - timing_internal;
        printf("\t\tCPU-GPU transfer : %lf ms\n", 1000*cpu_to_gpu_time);
        gpu_start = wtime();		

		//printf("Launching find_nearest_cluster Kernel with grid_size = %d, block_size = %d, shared_mem = %d KB\n", numClusterBlocks, numThreadsPerClusterBlock, clusterBlockSharedDataSize/1000);
        find_nearest_cluster
            <<< numClusterBlocks, numThreadsPerClusterBlock, clusterBlockSharedDataSize >>>
            (numCoords, numObjs, numClusters,
             deviceObjects, deviceClusters, deviceMembership, dev_delta_ptr);

        hipDeviceSynchronize(); checkLastCudaError();
		//printf("Kernels complete for itter %d, updating data in CPU\n", loop);
		
		gpu_time = wtime() - gpu_start;
        printf("\t\tGPU part: %lf ms\n", 1000*gpu_time);
        gpu_to_cpu_start = wtime();	
		
		/* TODO: Copy deviceMembership to membership
        checkCuda(hipMemcpy(...)); */
    	checkCuda(hipMemcpy(membership, deviceMembership,
                  numObjs*sizeof(int), hipMemcpyDeviceToHost));

    	/* TODO: Copy dev_delta_ptr to &delta
        checkCuda(hipMemcpy(...)); */
		checkCuda(hipMemcpy(&delta, dev_delta_ptr, sizeof(double), hipMemcpyDeviceToHost));
		gpu_to_cpu_time = wtime() - gpu_to_cpu_start;
        printf("\t\tGPU-CPU transfer : %lf ms\n", 1000*gpu_to_cpu_time);
		/* CPU part: Update cluster centers*/
  		cpu_start = wtime();

        for (i=0; i<numObjs; i++) {
            /* find the array index of nestest cluster center */
            index = membership[i];
			
            /* update new cluster centers : sum of objects located within */
            newClusterSize[index]++;
            for (j=0; j<numCoords; j++)
                newClusters[j][index] += objects[i*numCoords + j];
        }
 
        /* average the sum and replace old cluster centers with newClusters */
        for (i=0; i<numClusters; i++) {
            for (j=0; j<numCoords; j++) {
                if (newClusterSize[i] > 0)
                    dimClusters[j][i] = newClusters[j][i] / newClusterSize[i];
                newClusters[j][i] = 0.0;   /* set back to 0 */
            }
            newClusterSize[i] = 0;   /* set back to 0 */
        }

        delta /= numObjs;
       	//printf("delta is %f - ", delta);
        loop++; 
        //printf("completed loop %d\n", loop);
		cpu_time = wtime() - cpu_start;
		timing_internal = wtime() - timing_internal; 
		printf("\t\tCPU part : %lf ms\n", 1000*cpu_time);
		if ( timing_internal < timer_min) timer_min = timing_internal; 
		if ( timing_internal > timer_max) timer_max = timing_internal; 
	} while (delta > threshold && loop < loop_threshold);
    
    /*TODO: Update clusters using dimClusters. Be carefull of layout!!! clusters[numClusters][numCoords] vs dimClusters[numCoords][numClusters] */ 
	//  [numCoords][numObjs] layout
    for (i = 0; i < numCoords; i++) {
        for (j = 0; j < numClusters; j++) {
			clusters[j*numCoords+i] = dimClusters[i][j];
        }
    }	
    timing = wtime() - timing;
    printf("nloops = %d  : total = %lf ms\n\t-> t_loop_avg = %lf ms\n\t-> t_loop_min = %lf ms\n\t-> t_loop_max = %lf ms\n\n|-------------------------------------------|\n", 
    	loop, 1000*timing, 1000*timing/loop, 1000*timer_min, 1000*timer_max);

	char outfile_name[1024] = {0}; 
	sprintf(outfile_name, "Execution_logs/silver1-V100_Sz-%lu_Coo-%d_Cl-%d.csv", numObjs*numCoords*sizeof(double)/(1024*1024), numCoords, numClusters);
	FILE* fp = fopen(outfile_name, "a+");
	if(!fp) error("Filename %s did not open succesfully, no logging performed\n", outfile_name); 
	fprintf(fp, "%s,%d,%lf,%lf,%lf\n", "Transpose", blockSize, timing/loop, timer_min, timer_max);
	fclose(fp); 
	
    checkCuda(hipFree(deviceObjects));
    checkCuda(hipFree(deviceClusters));
    checkCuda(hipFree(deviceMembership));

    free(dimObjects[0]);
    free(dimObjects);
    free(dimClusters[0]);
    free(dimClusters);
    free(newClusters[0]);
    free(newClusters);
    free(newClusterSize);

    return;
}

