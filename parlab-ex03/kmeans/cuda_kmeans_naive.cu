#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "kmeans.h"
#include "alloc.h"
#include "error.h"

#ifdef __HIPCC__
inline void checkCuda(hipError_t e) {
    if (e != hipSuccess) {
        // hipGetErrorString() isn't always very helpful. Look up the error
        // number in the hipError_t enum in hip/driver_types.h in the CUDA includes
        // directory for a better explanation.
        error("CUDA Error %d: %s\n", e, hipGetErrorString(e));
    }
}

inline void checkLastCudaError() {
    checkCuda(hipGetLastError());
}
#endif

__device__ int get_tid() {
	/* TODO: Calculate 1-Dim global ID of a thread */
	return threadIdx.x + blockIdx.x * blockDim.x;
}

/* square of Euclid distance between two multi-dimensional points */
__host__ __device__ inline static
double euclid_dist_2(int    numCoords,
                    int    numObjs,
                    int    numClusters,
                    double *objects,     // [numObjs][numCoords]
                    double *clusters,    // [numClusters][numCoords]
                    int    objectId,
                    int    clusterId)
{
    int i;
    double ans=0.0;
    /* 
	for(i=0; i<numdims; i++)
        ans += (coord1[i]-coord2[i]) * (coord1[i]-coord2[i]);
    */
	/* TODO: Calculate the euclid_dist of elem=objectId of objects from elem=clusterId from clusters*/
    for(i = 0; i < numCoords; i++) {
		ans += (objects[objectId*numCoords+i] - clusters[clusterId*numCoords+i]) * (objects[objectId*numCoords+i] - clusters[clusterId*numCoords+i]);
    }
    return(ans);
}

__global__ static
void find_nearest_cluster(int numCoords,
                          int numObjs,
                          int numClusters
,                         double *objects,           //  [numObjs][numCoords]
                          double *deviceClusters,    //  [numClusters][numCoords]
                          int *deviceMembership,          //  [numObjs]
			  			  double *devdelta)
{

	/* Get the global ID of the thread. */
    int tid = get_tid(); 
    // int local_tid = threadIdx.x;
    // extern __shared__ double partial_devdelta[];
    // partial_devdelta[local_tid] = 0;
    /* TODO: Maybe something is missing here... should all threads run this? */
    // threads with tid > numObjs ??
    if (tid < numObjs) {
        int   index, i;
        double dist, min_dist;
	
        /* find the cluster id that has min distance to object */
        index = 0;
        /* TODO: call min_dist = euclid_dist_2(...) with correct objectId/clusterId */
	// objectId is tid maybe ?
	min_dist = euclid_dist_2(numCoords, numObjs, numClusters, objects, deviceClusters, tid, 0); 
        for (i=1; i<numClusters; i++) {
        	/* TODO: call dist = euclid_dist_2(...) with correct objectId/clusterId */
 	    	// again objectId is tid ? 
			dist = euclid_dist_2(numCoords, numObjs, numClusters, objects, deviceClusters, tid, i);
        	/* no need square root */
        	if (dist < min_dist) { /* find the min and its array index */
            	min_dist = dist;
            	index    = i;
        	}
        }

        if (deviceMembership[tid] != index) {
        	/* TODO: Maybe something is missing here... is this write safe? */
	    	// reduce cuda variable..how??
    	    // we will use shared memory for the reduction
	    	//partial_devdelta[local_tid] += 1.0;	    
	    	//(*devdelta)+= 1.0;
			atomicAdd(devdelta, 1.0);
        }
        /* assign the deviceMembership to object objectId */
        deviceMembership[tid] = index;
    }
	/*
    __syncthreads();
    int i = blockDim.x / 2;
    while(i != 0) {
        if(local_tid < i) {
            partial_devdelta[local_tid] += partial_devdelta[local_tid + i];
        }
        __syncthreads();
        i /= 2;
    }
    if(local_tid == 0) {
        atomicAdd(devdelta, partial_devdelta[0]);
    }
	*/
}

//
//  ----------------------------------------
//  DATA LAYOUT
//
//  objects         [numObjs][numCoords]
//  clusters        [numClusters][numCoords]
//  newClusters     [numClusters][numCoords]
//  deviceObjects   [numObjs][numCoords]
//  deviceClusters  [numClusters][numCoords]
//  ----------------------------------------
//
/* return an array of cluster centers of size [numClusters][numCoords]       */            
void kmeans_gpu(	double *objects,      /* in: [numObjs][numCoords] */
		               	int     numCoords,    /* no. features */
		               	int     numObjs,      /* no. objects */
		               	int     numClusters,  /* no. clusters */
		               	double   threshold,    /* % objects change membership */
		               	long    loop_threshold,   /* maximum number of iterations */
		               	int    *membership,   /* out: [numObjs] */
				double * clusters,   /* out: [numClusters][numCoords] */
				int blockSize)
{
    double timing = wtime(), timing_internal, timer_min = 1e42, timer_max = 0;
    //int    loop_iterations = 0; 
    int      i, j, index, loop=0;
    int     *newClusterSize; /* [numClusters]: no. objects assigned in each
                                new cluster */
    double  delta = 0, *dev_delta_ptr;          /* % of objects change their clusters */
    double  **newClusters = (double**) calloc_2d(numClusters, numCoords, sizeof(double));
    
    double *deviceObjects;
    double *deviceClusters;
    int *deviceMembership;

    printf("\n|-------------Naive GPU Kmeans--------------|\n\n");

    /* initialize membership[] */
    for (i=0; i<numObjs; i++) membership[i] = -1;

    /* need to initialize newClusterSize and newClusters[0] to all 0 */
    newClusterSize = (int*) calloc(numClusters, sizeof(int));
    assert(newClusterSize != NULL); 
    
    timing = wtime() - timing;
    printf("t_alloc: %lf ms\n\n", 1000*timing);
    timing = wtime(); 

    const unsigned int numThreadsPerClusterBlock = (numObjs > blockSize)? blockSize: numObjs;
    const unsigned int numClusterBlocks = numObjs/blockSize + 1; /* TODO: Calculate Grid size, e.g. number of blocks. */
    const unsigned int clusterBlockSharedDataSize = numThreadsPerClusterBlock*sizeof(double);
       
    checkCuda(hipMalloc(&deviceObjects, numObjs*numCoords*sizeof(double)));
    checkCuda(hipMalloc(&deviceClusters, numClusters*numCoords*sizeof(double)));
    checkCuda(hipMalloc(&deviceMembership, numObjs*sizeof(int)));
    checkCuda(hipMalloc(&dev_delta_ptr, sizeof(double)));
    
    timing = wtime() - timing;
    printf("t_alloc_gpu: %lf ms\n\n", 1000*timing);
    timing = wtime(); 
        
    checkCuda(hipMemcpy(deviceObjects, objects,
              numObjs*numCoords*sizeof(double), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(deviceMembership, membership,
              numObjs*sizeof(int), hipMemcpyHostToDevice));
    timing = wtime() - timing;
    printf("t_get_gpu: %lf ms\n\n", 1000*timing);
    timing = wtime();   
   	double cpu_to_gpu_time, gpu_start, gpu_time,
        gpu_to_cpu_start, gpu_to_cpu_time, cpu_start, cpu_time;
 
    do {
        timing_internal = wtime(); 
 
		/* GPU part: calculate new memberships */
		        
        /* TODO: Copy clusters to deviceClusters
        checkCuda(hipMemcpy(...)); */	
        checkCuda(hipMemcpy(deviceClusters, clusters,
		  numClusters*numCoords*sizeof(double), hipMemcpyHostToDevice));
        checkCuda(hipMemset(dev_delta_ptr, 0, sizeof(double)));
			
		cpu_to_gpu_time = wtime() - timing_internal;
        printf("\t\tCPU-GPU transfer : %lf ms\n", 1000*cpu_to_gpu_time);
        gpu_start = wtime();		
          
		//printf("Launching find_nearest_cluster Kernel with grid_size = %d, block_size = %d, shared_mem = %d KB\n", numClusterBlocks, numThreadsPerClusterBlock, clusterBlockSharedDataSize/1000);
        find_nearest_cluster
            <<< numClusterBlocks, numThreadsPerClusterBlock, clusterBlockSharedDataSize >>>
            (numCoords, numObjs, numClusters,
             deviceObjects, deviceClusters, deviceMembership, dev_delta_ptr);

        hipDeviceSynchronize(); checkLastCudaError();
		gpu_time = wtime() - gpu_start;
    	printf("\t\tGPU part: %lf ms\n", 1000*gpu_time);
    	gpu_to_cpu_start = wtime();
	
		//printf("Kernels complete for itter %d, updating data in CPU\n", loop);
		
		/* TODO: Copy deviceMembership to membership
        checkCuda(hipMemcpy(...)); */
    	checkCuda(hipMemcpy(membership, deviceMembership,
		  numObjs*sizeof(int), hipMemcpyDeviceToHost));
    	/* TODO: Copy dev_delta_ptr to &delta
        checkCuda(hipMemcpy(...)); */
		checkCuda(hipMemcpy(&delta, dev_delta_ptr, sizeof(double), hipMemcpyDeviceToHost));
		
		gpu_to_cpu_time = wtime() - gpu_to_cpu_start;
        printf("\t\tGPU-CPU transfer : %lf ms\n", 1000*gpu_to_cpu_time);
        /* CPU part: Update cluster centers*/
        cpu_start = wtime();
				
        for (i=0; i<numObjs; i++) {
            /* find the array index of nestest cluster center */
            index = membership[i];
			
            /* update new cluster centers : sum of objects located within */
            newClusterSize[index]++;
            for (j=0; j<numCoords; j++)
                newClusters[index][j] += objects[i*numCoords + j];
        }
        
        /* average the sum and replace old cluster centers with newClusters */
        for (i=0; i<numClusters; i++) {
            for (j=0; j<numCoords; j++) {
                if (newClusterSize[i] > 0)
                    clusters[i*numCoords + j] = newClusters[i][j] / newClusterSize[i];
                newClusters[i][j] = 0.0;   /* set back to 0 */
            }
            newClusterSize[i] = 0;   /* set back to 0 */
        }
        delta /= numObjs;
       	//printf("delta is %f - ", delta);
        loop++; 
        //printf("completed loop %d\n", loop);   
		cpu_time = wtime() - cpu_start;
		timing_internal = wtime() - timing_internal; 
		printf("\t\tCPU part : %lf ms\n", 1000*cpu_time);
		if ( timing_internal < timer_min) timer_min = timing_internal; 
		if ( timing_internal > timer_max) timer_max = timing_internal;      
    } while (delta > threshold && loop < loop_threshold);
    
    timing = wtime() - timing;
    printf("nloops = %d  : total = %lf ms\n\t-> t_loop_avg = %lf ms\n\t-> t_loop_min = %lf ms\n\t-> t_loop_max = %lf ms\n\n|-------------------------------------------|\n", 
    	loop, 1000*timing, 1000*timing/loop, 1000*timer_min, 1000*timer_max);

	char outfile_name[1024] = {0}; 
	sprintf(outfile_name, "Execution_logs/silver1-V100_Sz-%lu_Coo-%d_Cl-%d.csv", numObjs*numCoords*sizeof(double)/(1024*1024), numCoords, numClusters);
	FILE* fp = fopen(outfile_name, "a+");
	if(!fp) error("Filename %s did not open succesfully, no logging performed\n", outfile_name); 
	fprintf(fp, "%s,%d,%lf,%lf,%lf\n", "Naive", blockSize, timing/loop, timer_min, timer_max);
	fclose(fp); 
    checkCuda(hipFree(deviceObjects));
    checkCuda(hipFree(deviceClusters));
    checkCuda(hipFree(deviceMembership));

    free(newClusters[0]);
    free(newClusters);
    free(newClusterSize);

    return;
}

